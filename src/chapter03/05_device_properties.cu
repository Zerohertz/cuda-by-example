#include <iostream>

#include "../common/util.h"

using namespace std;

int main(void)
{
    hipDeviceProp_t prop;
    int            dev;

    hipGetDevice(&dev);
    cout << "ID of current CUDA device: " << dev << endl;

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 3;
    hipChooseDevice(&dev, &prop);
    cout << "ID of CUDA device closet to version 1.3: " << dev << endl;

    hipSetDevice(dev);

    CHECK_CUDA_ERROR();
    return 0;
}

/*
 * ID of current CUDA device: 0
 * ID of CUDA device closet to version 1.3: 0
 */
