#include "../../include/handler.cuh"

int main(void)
{
    hipDeviceProp_t prop;
    int            dev;

    CUDA_CHECK(hipGetDevice(&dev));
    LOG_INFO("ID of current CUDA device: ", dev);

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 3;
    CUDA_CHECK(hipChooseDevice(&dev, &prop));
    LOG_INFO("ID of CUDA device closet to version 1.3: ", dev);

    CUDA_CHECK(hipSetDevice(dev));
    return 0;
}

/*
 * [2025-09-03 20:54:42] [src/chapter03/05_device_properties.cu:8] ✅ hipGetDevice(&dev)
 * [2025-09-03 20:54:42] [src/chapter03/05_device_properties.cu:9] ℹ️ ID of current CUDA device: 0
 * [2025-09-03 20:54:42] [src/chapter03/05_device_properties.cu:14] ✅ hipChooseDevice(&dev, &prop)
 * [2025-09-03 20:54:42] [src/chapter03/05_device_properties.cu:15] ℹ️ ID of CUDA device closet to version 1.3: 0
 * [2025-09-03 20:54:43] [src/chapter03/05_device_properties.cu:17] ✅ hipSetDevice(dev)
 */
