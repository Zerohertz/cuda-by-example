#include "hip/hip_runtime.h"
#include "../../include/handler.cuh"

#define N 1024

__global__ void kernel(hipTextureObject_t tex)
{
    int   tid = blockIdx.x * blockDim.x + threadIdx.x;
    float x   = tex1Dfetch<float>(tex, tid);
}

void call_kernel(hipTextureObject_t tex) { kernel<<<1, 256>>>(tex); }

int main()
{
    float *buffer;
    float  value = 023323.0f;
    buffer       = &value;
    CUDA_CHECK(hipMalloc(&buffer, N * sizeof(float)));

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType                = hipResourceTypeLinear;
    resDesc.res.linear.devPtr      = buffer;
    resDesc.res.linear.desc.f      = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x      = 32;
    resDesc.res.linear.sizeInBytes = N * sizeof(float);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;

    hipTextureObject_t tex = 0;
    CUDA_CHECK(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

    call_kernel(tex);

    CUDA_CHECK(hipDestroyTextureObject(tex));
    CUDA_CHECK(hipFree(buffer));

    return 0;
}

/*
 * [2025-09-23 00:27:40] [src/lecture/chapter04/08_texture.cu:18] ✅ hipMalloc(&buffer, N * sizeof(float))
 * [2025-09-23 00:27:40] [src/lecture/chapter04/08_texture.cu:33] ✅ hipCreateTextureObject(&tex, &resDesc, &texDesc,
 * NULL)
 * [2025-09-23 00:27:40] [src/lecture/chapter04/08_texture.cu:37] ✅ hipDestroyTextureObject(tex)
 * [2025-09-23 00:27:40] [src/lecture/chapter04/08_texture.cu:38] ✅ hipFree(buffer)
 */
